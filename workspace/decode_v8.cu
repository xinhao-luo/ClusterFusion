#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"                
#include "hip/hip_cooperative_groups.h"
#include "hip/hip_fp16.h"
#include <hip/hip_runtime.h>    
#include <cuda/barrier>
#include <cudaTypedefs.h>
using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;
#include <iostream>
#include <random>
#include <stdio.h>

// nvcc --generate-code=arch=compute_90a,code=sm_90a -O3 -std=c++17 -lcuda decode_v8.cu -o test && ./test

#define HEAD_DIM 128    
#define HEAD_NUM 32     
#define FFN_DIM 12288   
#define HIDDEN_DIM 4096 
#define SEQ_LEN 16384

#define NUM_WARPS 4 // 4 8 16 32
#define WARP_SIZE 32
#define BLOCK_SIZE (NUM_WARPS * WARP_SIZE) 
#define CLUSTER_SIZE 4 // 2 4
#define NUM_PER_THREAD 8
#define NUM_ROW_PER_WARP (HEAD_DIM / NUM_WARPS) 
#define NUM_THREAD_PER_ROW (WARP_SIZE / NUM_ROW_PER_WARP) 
#define NUM_PER_ROW (NUM_PER_THREAD * NUM_THREAD_PER_ROW) 
#define DIM_PER_BLOCK (HIDDEN_DIM / CLUSTER_SIZE)
#define KV_DIM_PER_BLOCK (SEQ_LEN / CLUSTER_SIZE) 
#define FFN_DIM_PER_CLUSTER (FFN_DIM / HEAD_NUM) 
#define MAX_SMEM_DIM FFN_DIM_PER_CLUSTER

#define TMA_LOAD_ONCE 64 // 8 16 32 64 128 256
#define TMA_LOAD_ONCE_MAX 256
#define TMA_LOAD_ONCE_NUM (TMA_LOAD_ONCE * HEAD_DIM)
#define TMA_LOAD_ONCE_SIZE (TMA_LOAD_ONCE_NUM * sizeof(half))
#define TMA_LOAD_ONCE_ATTN (TMA_LOAD_ONCE / 2)
#define TMA_LOAD_ONCE_NUM_ATTN ((TMA_LOAD_ONCE * HEAD_DIM) / 2)
#define TMA_LOAD_ONCE_SIZE_ATTN (TMA_LOAD_ONCE_NUM_ATTN * sizeof(half))
#define TMA_LOAD_ONCE_NUM_FFN (TMA_LOAD_ONCE * TMA_LOAD_ONCE_MAX)
#define TMA_LOAD_ONCE_NUM_FFN_TOTAL (TMA_LOAD_ONCE * FFN_DIM_PER_CLUSTER)
#define TMA_LOAD_ONCE_SIZE_FFN (TMA_LOAD_ONCE_NUM_FFN_TOTAL * sizeof(half))

#define NUM_THREAD_PER_ROW_2 (HEAD_DIM / NUM_PER_THREAD) // 16
#define NUM_ROW_PER_WARP_2 (WARP_SIZE / NUM_THREAD_PER_ROW_2) // 2
#define NUM_PER_ROW_2 (NUM_WARPS * NUM_ROW_PER_WARP_2) // 8
#define DEC_TILE (TMA_LOAD_ONCE_ATTN / NUM_PER_ROW_2)
#define NUM_ROW_PER_WARP_3 (TMA_LOAD_ONCE / NUM_WARPS) 
#define NUM_THREAD_PER_ROW_3 (WARP_SIZE / NUM_ROW_PER_WARP_3) 
#define NUM_PER_ROW_3 (NUM_PER_THREAD * NUM_THREAD_PER_ROW_3) 

template <typename T>
void fill_matrix(T* mat, int sz) {
    std::random_device r;
    std::mt19937 rng(r());
    std::normal_distribution<float> norm_dist(0.0, 5.0);
    for (int i = 0; i < sz; i++) {
        if constexpr(std::is_same<T, half>::value) {
            mat[i] = __float2half(0.01f);
        }   
    }   
}

__global__ void __cluster_dims__(CLUSTER_SIZE, 1, 1) single_decode(
    half* output, // 1 * hidden_dim
    half* input,  // 1 * hidden_dim
    half* global_reduce,    // hidden_dim  
    half* w_rms_input,// hidden_dim
    half* w_rms_attn, // hidden_dim
    float* cos,       // head_dim
    float* sin,       // head_dim
    const __grid_constant__ CUtensorMap tensor_map, // 3 * hidden_dim * hidden_dim
    const __grid_constant__ CUtensorMap tensor_map_k_cache, // seqlen * head_num * head_dim
    const __grid_constant__ CUtensorMap tensor_map_v_cache, // seqlen * head_num * head_dim
    const __grid_constant__ CUtensorMap tensor_map_weight_o, // hidden_dim * hidden_dim
    const __grid_constant__ CUtensorMap tensor_map_weight_gate_up, // 2 * hidden_dim * ffn_dim
    const __grid_constant__ CUtensorMap tensor_map_weight_gate_up_,// 2 * hidden_dim * ffn_dim
    const __grid_constant__ CUtensorMap tensor_map_weight_down, // ffn_dim * hidden_dim
    const __grid_constant__ CUtensorMap tensor_map_weight_down_ // ffn_dim * hidden_dim
)
{
    namespace cg = cooperative_groups;
    cg::grid_group grid             = cg::this_grid();
    cg::cluster_group cluster       = cg::this_cluster();
    cg::thread_block block          = cg::this_thread_block();
    const uint32_t head_id          = grid.cluster_rank() % HEAD_NUM;
    const uint32_t cluster_block_id = cluster.block_rank();
    const uint32_t tid              = block.thread_rank();
    const uint32_t lane_id = tid % WARP_SIZE; 
    const uint32_t warp_id = tid / WARP_SIZE;
    const uint32_t tile_row = tid / NUM_THREAD_PER_ROW_2;
    const uint32_t tile_col = tid % NUM_THREAD_PER_ROW_2;

    // Init shared memory
    __shared__ __align__(16) half input_shmem[DIM_PER_BLOCK];
    __shared__ float reduction[NUM_WARPS];
    __shared__ float cluster_local_sum;
    __shared__ alignas(128) half weight[2 * TMA_LOAD_ONCE * MAX_SMEM_DIM];
    __shared__ __align__(16) half local_qkv[MAX_SMEM_DIM + MAX_SMEM_DIM + HEAD_DIM];
    __shared__ __align__(16) half local_output[HEAD_DIM];

    // Init register
    float local_sum = 0;
    half __align__(16) reg_input_norm[2], reg_weight_norm[2];
    float tmp = 0.0;
    half __align__(16) reg_input[NUM_PER_THREAD];
    half __align__(16) reg_weight[NUM_PER_THREAD];
    half2 q_rope, q_rope_1;
    half2 k_rope, k_rope_1;
    float2 cos_reg, sin_reg;
    uint32_t size;
    half2 buffer;
    half __align__(16) reg_reduce[NUM_PER_THREAD];
    float __align__(16) qk[DEC_TILE];
    float tmp_ffn[FFN_DIM_PER_CLUSTER / HEAD_DIM];
    for (int j = 0; j < FFN_DIM_PER_CLUSTER / HEAD_DIM; j++){
      tmp_ffn[j] = 0.0;
    }

    // Init barrier
    #pragma nv_diag_suppress static_var_with_dynamic_init
    __shared__ barrier bar[4];
    barrier::arrival_token token[4];
    __shared__ uint64_t barrier;
    uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(&barrier));
    if (tid == 0) {
        init(&bar[0], blockDim.x);
        cde::fence_proxy_async_shared_cta();
        init(&bar[1], blockDim.x);
        cde::fence_proxy_async_shared_cta();
        init(&bar[2], blockDim.x);
        cde::fence_proxy_async_shared_cta();
        init(&bar[3], blockDim.x);
        cde::fence_proxy_async_shared_cta();
    }
    block.sync();

    // Precompute some indices
    uint cluster_block_st_id = cluster_block_id * DIM_PER_BLOCK;
    uint input_idx = (lane_id % NUM_THREAD_PER_ROW) * NUM_PER_THREAD;
    uint weight_idx = warp_id * NUM_ROW_PER_WARP + lane_id / NUM_THREAD_PER_ROW;
    uint input_idx_2 = (lane_id % NUM_THREAD_PER_ROW_2) * NUM_PER_THREAD;
    uint weight_idx_2 = warp_id * NUM_PER_ROW_2 + (lane_id / NUM_THREAD_PER_ROW_2) * DEC_TILE;
    uint input_idx_3 = (lane_id % NUM_THREAD_PER_ROW_3) * NUM_PER_THREAD;
    uint weight_idx_3 = warp_id * NUM_ROW_PER_WARP_3 + lane_id / NUM_THREAD_PER_ROW_3;

    // Load input to shared memory
    #pragma unroll
    for (int i = tid * 8; i < DIM_PER_BLOCK; i+=BLOCK_SIZE * 8) {
        *(uint4*)(&input_shmem[i]) = *(uint4*)(&input[cluster_block_st_id + i]);
    }
    block.sync();

    // RMSNorm
    for (int d = tid * 2; d < DIM_PER_BLOCK; d+=BLOCK_SIZE * 2) { 
        *(half2*)(&reg_input_norm[0]) = *(half2*)(&input_shmem[d]);
        for (int di = 0; di < 2; di++)
            local_sum += __half2float(reg_input_norm[di] * reg_input_norm[di]);
    }
    #pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        local_sum += __shfl_down_sync(0xffffffff, local_sum, mask);
    }
    if (lane_id == 0){
        reduction[warp_id] = local_sum;
    }
    block.sync(); 
    if (tid < NUM_WARPS) 
        local_sum = reduction[tid];
    #pragma unroll
    for (int mask = NUM_WARPS / 2; mask > 0; mask >>= 1) {
        local_sum += __shfl_down_sync(0xffffffff, local_sum, mask);
    } 
    if (tid == 0)
        cluster_local_sum = local_sum;
    cluster.sync();
    // Reduce through DSM
    for (int i = 1; i < cluster.num_blocks() - 1; i++) {
        if (tid == 0) {
            local_sum = cluster_local_sum;
            int dst_cta = (cluster_block_id + i) % cluster.num_blocks();
            float* dst_shmem = cluster.map_shared_rank(&cluster_local_sum, dst_cta);
            atomicAdd(dst_shmem, local_sum);
        }
        cluster.sync();
    }
    float eps = 1e-6;
    half rms_rcp = __float2half(1.f / (std::sqrt(cluster_local_sum / float(HIDDEN_DIM)) + eps));
    for (int d = tid * 2; d < DIM_PER_BLOCK; d+=BLOCK_SIZE * 2) { 
        *(half2*)(&reg_input_norm[0]) = *(half2*)(&input_shmem[d]);
        *(half2*)(&reg_input_norm[0]) = __hmul2(*(half2*)(&reg_input_norm[0]), {rms_rcp, rms_rcp});
        *(half2*)(&reg_weight_norm[0]) = *(half2*)(&w_rms_input[cluster_block_st_id + d]);
        *(half2*)(&input_shmem[d]) = __hmul2(*(half2*)(&reg_input_norm[0]), *(half2*)(&reg_weight_norm[0]));
    }
    block.sync();

    // Compute input @ w_q
    // Preload weight_q
    if (tid == 0) {
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[0], &tensor_map, head_id * HEAD_DIM, cluster_block_st_id, bar[0]);
        token[0] = cuda::device::barrier_arrive_tx(bar[0], 1, TMA_LOAD_ONCE_SIZE);
    } else {
        token[0] = bar[0].arrive();
    }

    for (int id = 1; id < DIM_PER_BLOCK / TMA_LOAD_ONCE; id++) {
        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM], &tensor_map, head_id * HEAD_DIM, cluster_block_st_id + id * TMA_LOAD_ONCE, bar[id % 2]);
            token[id % 2] = cuda::device::barrier_arrive_tx(bar[id % 2], 1, TMA_LOAD_ONCE_SIZE);
        } else {
            token[id % 2] = bar[id % 2].arrive();
        }
        bar[(id - 1) % 2].wait(std::move(token[(id - 1) % 2]));
        for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
            *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + (id - 1) * TMA_LOAD_ONCE + i]);
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp += __half2float(reg_input[d] * weight[((id - 1) % 2) * TMA_LOAD_ONCE_NUM + (input_idx + i + d) * HEAD_DIM + weight_idx]);
            }
        }
    }
    bar[1].wait(std::move(token[1]));
    for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
        *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + ((DIM_PER_BLOCK / TMA_LOAD_ONCE) - 1) * TMA_LOAD_ONCE + i]);
        #pragma unroll
        for (int d = 0; d < NUM_PER_THREAD; d++) {
            tmp += __half2float(reg_input[d] * weight[TMA_LOAD_ONCE_NUM + (input_idx + i + d) * HEAD_DIM + weight_idx]);
        }
    }
    #pragma unroll
    for (int mask = (NUM_THREAD_PER_ROW >> 1); mask > 0; mask >>= 1) {
        tmp += __shfl_down_sync(0xffffffff, tmp, mask);
    }
    if (lane_id % NUM_THREAD_PER_ROW == 0) {
        local_qkv[warp_id * NUM_ROW_PER_WARP + lane_id / NUM_THREAD_PER_ROW] = __float2half(tmp);
    }

    // Compute input @ w_k
    // Preload weight_k
    tmp = 0.0;
    if (tid == 0) {
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[0], &tensor_map, head_id * HEAD_DIM, HIDDEN_DIM + cluster_block_st_id, bar[0]);
        token[0] = cuda::device::barrier_arrive_tx(bar[0], 1, TMA_LOAD_ONCE_SIZE);
    } else {
        token[0] = bar[0].arrive();
    }

    for (int id = 1; id < DIM_PER_BLOCK / TMA_LOAD_ONCE; id++) {
        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM], &tensor_map, head_id * HEAD_DIM, HIDDEN_DIM + cluster_block_st_id + id * TMA_LOAD_ONCE, bar[id % 2]);
            token[id % 2] = cuda::device::barrier_arrive_tx(bar[id % 2], 1, TMA_LOAD_ONCE_SIZE);
        } else {
            token[id % 2] = bar[id % 2].arrive();
        }
        bar[(id - 1) % 2].wait(std::move(token[(id - 1) % 2]));
        for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
            *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + (id - 1) * TMA_LOAD_ONCE + i]);
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp += __half2float(reg_input[d] * weight[((id - 1) % 2) * TMA_LOAD_ONCE_NUM + (input_idx + i + d) * HEAD_DIM + weight_idx]);
            }
        }
    }
    bar[1].wait(std::move(token[1]));
    for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
        *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + ((DIM_PER_BLOCK / TMA_LOAD_ONCE) - 1) * TMA_LOAD_ONCE + i]);
        #pragma unroll
        for (int d = 0; d < NUM_PER_THREAD; d++) {
            tmp += __half2float(reg_input[d] * weight[TMA_LOAD_ONCE_NUM + (input_idx + i + d) * HEAD_DIM + weight_idx]);
        }
    }
    #pragma unroll
    for (int mask = (NUM_THREAD_PER_ROW >> 1); mask > 0; mask >>= 1) {
        tmp += __shfl_down_sync(0xffffffff, tmp, mask);
    }
    if (lane_id % NUM_THREAD_PER_ROW == 0) {
        local_qkv[HEAD_DIM + warp_id * NUM_ROW_PER_WARP + lane_id / NUM_THREAD_PER_ROW] = __float2half(tmp);
    }
    
    // Compute input @ w_v
    // Preload weight_v
    tmp = 0.0;
    if (tid == 0) {
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[0], &tensor_map, head_id * HEAD_DIM, HIDDEN_DIM * 2 + cluster_block_st_id, bar[0]);
        token[0] = cuda::device::barrier_arrive_tx(bar[0], 1, TMA_LOAD_ONCE_SIZE);
    } else {
        token[0] = bar[0].arrive();
    }

    for (int id = 1; id < DIM_PER_BLOCK / TMA_LOAD_ONCE; id++) {
        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM], &tensor_map, head_id * HEAD_DIM, HIDDEN_DIM * 2 + cluster_block_st_id + id * TMA_LOAD_ONCE, bar[id % 2]);
            token[id % 2] = cuda::device::barrier_arrive_tx(bar[id % 2], 1, TMA_LOAD_ONCE_SIZE);
        } else {
            token[id % 2] = bar[id % 2].arrive();
        }
        bar[(id - 1) % 2].wait(std::move(token[(id - 1) % 2]));
        for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
            *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + (id - 1) * TMA_LOAD_ONCE + i]);
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp += __half2float(reg_input[d] * weight[((id - 1) % 2) * TMA_LOAD_ONCE_NUM + (input_idx + i + d) * HEAD_DIM + weight_idx]);
            }
        }
    }
    bar[1].wait(std::move(token[1]));
    for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
        *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + ((DIM_PER_BLOCK / TMA_LOAD_ONCE) - 1) * TMA_LOAD_ONCE + i]);
        #pragma unroll
        for (int d = 0; d < NUM_PER_THREAD; d++) {
            tmp += __half2float(reg_input[d] * weight[TMA_LOAD_ONCE_NUM + (input_idx + i + d) * HEAD_DIM + weight_idx]);
        }
    }
    #pragma unroll
    for (int mask = (NUM_THREAD_PER_ROW >> 1); mask > 0; mask >>= 1) {
        tmp += __shfl_down_sync(0xffffffff, tmp, mask);
    }
    if (lane_id % NUM_THREAD_PER_ROW == 0) {
        local_qkv[HEAD_DIM * 2 + warp_id * NUM_ROW_PER_WARP + lane_id / NUM_THREAD_PER_ROW] = __float2half(tmp);
    }

    // DSM Ring-All reduce
    size = (HEAD_DIM * 3) * sizeof(half);
    for (int i = 1; i < cluster.num_blocks() - 1; i++) {
        if (tid == 0) {
            asm volatile (
                "mbarrier.init.shared::cta.b64 [%0], %1;"
                :
                : "r"(bar_ptr), "r"(1)
            );
            asm volatile (
                "mbarrier.arrive.expect_tx.shared::cta.b64 _, [%0], %1;"
                :
                : "r"(bar_ptr), "r"(size)
            );
        }
        cluster.sync();
        if (tid == 0) {
            uint32_t src_addr = static_cast<uint32_t>(__cvta_generic_to_shared(local_qkv));
            uint32_t dst_addr = static_cast<uint32_t>(__cvta_generic_to_shared(weight));
            uint32_t dst_cta = (cluster_block_id + i) % cluster.num_blocks();
            uint32_t neighbor_dst_addr;
            asm volatile (
                "mapa.shared::cluster.u32 %0, %1, %2;\n"
                : "=r"(neighbor_dst_addr)
                : "r"(dst_addr), "r"(dst_cta)
            );
            uint32_t neighbor_dst_bar;
            asm volatile (
                "mapa.shared::cluster.u32 %0, %1, %2;\n"
                : "=r"(neighbor_dst_bar)
                : "r"(bar_ptr), "r"(dst_cta)
            );
            asm volatile (
                "cp.async.bulk.shared::cluster.shared::cta.mbarrier::complete_tx::bytes [%0], [%1], %2, [%3];"
                :
                :"r"(neighbor_dst_addr), "r"(src_addr), "r"(size), "r"(neighbor_dst_bar)
                : "memory"
            );
        }
        asm volatile (
            "{\n"
            ".reg .pred                P1;\n"
            "LAB_WAIT:\n"
            "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1;\n"
            "@P1                       bra.uni DONE;\n"
            "bra.uni                   LAB_WAIT;\n"
            "DONE:\n"
            "}\n"
            :: "r"(bar_ptr),
            "r"(0)
        );

        // Local reduce-add
        if (tid < HEAD_DIM / 2) {
            for (int j = 0; j < 3; j++) {
                buffer = *(half2*)(&weight[j * HEAD_DIM + tid * 2]);
                *(half2*)(&local_qkv[j * HEAD_DIM + tid * 2]) = __hadd2(*(half2*)(&local_qkv[j * HEAD_DIM + tid * 2]), buffer);
            }
        }
        cluster.sync();
    }
    // if(head_id == 0 && cluster_block_id == 0 && tid == 0)
    //     printf("%f, %f, %f, %f, %f, %f \n", __half2float(local_qkv[0]), __half2float(local_qkv[127]), __half2float(local_qkv[HEAD_DIM + 0]), __half2float(local_qkv[HEAD_DIM + 127]), __half2float(local_qkv[HEAD_DIM * 2 + 0]), __half2float(local_qkv[HEAD_DIM * 2 + 127]));

    // Compute RoPE
    if (tid < HEAD_DIM / 2) {
        q_rope = *(half2*)(&local_qkv[tid * 2]);
        k_rope = *(half2*)(&local_qkv[HEAD_DIM + tid * 2]);
        if (tid * 2 < HEAD_DIM / 2) {
            q_rope_1 = *(half2*)(&local_qkv[HEAD_DIM / 2 + tid * 2]);
            k_rope_1 = *(half2*)(&local_qkv[HEAD_DIM + HEAD_DIM / 2 + tid * 2]);
            cos_reg = {cos[tid * 2], cos[tid * 2 + 1]};
            sin_reg = {-sin[HEAD_DIM / 2 + tid * 2], -sin[HEAD_DIM / 2 + tid * 2 + 1]};
        } else {
            q_rope_1 = *(half2*)(&local_qkv[tid * 2 - HEAD_DIM / 2]);
            k_rope_1 = *(half2*)(&local_qkv[HEAD_DIM + tid * 2 - HEAD_DIM / 2]);
            cos_reg = {cos[tid * 2], cos[tid * 2 + 1]};
            sin_reg = {sin[tid * 2 - HEAD_DIM / 2], sin[tid * 2 + 1 - HEAD_DIM / 2]};
        }
        *(half2*)(&local_qkv[tid * 2]) = __hadd2(__hmul2(q_rope, __float22half2_rn(cos_reg)), __hmul2(q_rope_1, __float22half2_rn(sin_reg)));
        *(half2*)(&local_qkv[HEAD_DIM + tid * 2]) = __hadd2(__hmul2(k_rope, __float22half2_rn(cos_reg)), __hmul2(k_rope_1, __float22half2_rn(sin_reg)));
    }

    // Compute flash-decoding
    local_sum = 0.0f;
    if(lane_id == 0)
        reduction[warp_id] = 0.0f;
    for(int i = 0; i < NUM_PER_THREAD; i++)
        reg_reduce[i] = __float2half(0.0f);
    *(uint4*)(&reg_input[0]) = *(uint4*)(&local_qkv[input_idx_2]);
    block.sync();

    // Preload kv_cache
    if (tid == 0) {
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[0], &tensor_map_k_cache, head_id * HEAD_DIM, cluster_block_id * KV_DIM_PER_BLOCK, bar[0]);
        token[0] = cuda::device::barrier_arrive_tx(bar[0], 1, TMA_LOAD_ONCE_SIZE_ATTN);
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[TMA_LOAD_ONCE_NUM_ATTN], &tensor_map_v_cache, head_id * HEAD_DIM, cluster_block_id * KV_DIM_PER_BLOCK, bar[2]);
        token[2] = cuda::device::barrier_arrive_tx(bar[2], 1, TMA_LOAD_ONCE_SIZE_ATTN);
    } else {
        token[0] = bar[0].arrive();
        token[2] = bar[2].arrive();
    }

    for (int id = 1; id < KV_DIM_PER_BLOCK / TMA_LOAD_ONCE_ATTN; id++) {
        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM], &tensor_map_k_cache, head_id * HEAD_DIM, cluster_block_st_id + id * TMA_LOAD_ONCE_ATTN, bar[id % 2]);
            token[id % 2] = cuda::device::barrier_arrive_tx(bar[id % 2], 1, TMA_LOAD_ONCE_SIZE_ATTN);
        } else {
            token[id % 2] = bar[id % 2].arrive();
        }
        bar[(id - 1) % 2].wait(std::move(token[(id - 1) % 2]));
        for (int j = 0; j < DEC_TILE; j++) {
            *(uint4*)(&reg_weight[0]) = *(uint4*)(&weight[((id - 1) % 2) * TMA_LOAD_ONCE_NUM + (weight_idx_2 + j) * HEAD_DIM + input_idx_2]);
            qk[j] = 0.0f;
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                qk[j] += __half2float(reg_input[d] * reg_weight[d]);
            }
            #pragma unroll
            for (int mask = (NUM_THREAD_PER_ROW_2 >> 1); mask > 0; mask >>= 1) {
                qk[j] += __shfl_down_sync(0xffffffff, qk[j], mask);
            }
            qk[j] = __expf(qk[j] * __frsqrt_rn(HEAD_DIM));
            local_sum += qk[j];
        }

        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM + TMA_LOAD_ONCE_NUM_ATTN], &tensor_map_v_cache, head_id * HEAD_DIM, cluster_block_st_id + id * TMA_LOAD_ONCE_ATTN, bar[2 + id % 2]);
            token[2 + id % 2] = cuda::device::barrier_arrive_tx(bar[2 + id % 2], 1, TMA_LOAD_ONCE_SIZE_ATTN);
        } else {
            token[2 + id % 2] = bar[2 + id % 2].arrive();
        }
        bar[2 + (id - 1) % 2].wait(std::move(token[2 + (id - 1) % 2]));
        for (int j = 0; j < DEC_TILE; j++) {
            *(uint4*)(&reg_weight[0]) = *(uint4*)(&weight[((id - 1) % 2) * TMA_LOAD_ONCE_NUM + TMA_LOAD_ONCE_NUM_ATTN + (weight_idx_2 + j) * HEAD_DIM + input_idx_2]);
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                reg_reduce[d] += __float2half(qk[j] * __half2float(reg_weight[d]));
            }
        }
    }
    bar[1].wait(std::move(token[1]));
    for (int j = 0; j < DEC_TILE; j++) {
        if (cluster_block_id == CLUSTER_SIZE - 1 && warp_id == NUM_WARPS - 1 && lane_id / NUM_THREAD_PER_ROW_2 == 1 && j == DEC_TILE - 1)
            *(uint4*)(&reg_weight[0]) = *(uint4*)(&local_qkv[HEAD_DIM + input_idx_2]);
        else
            *(uint4*)(&reg_weight[0]) = *(uint4*)(&weight[TMA_LOAD_ONCE_NUM + (weight_idx_2 + j) * HEAD_DIM + input_idx_2]);
        qk[j] = 0.0f;
        #pragma unroll
        for (int d = 0; d < NUM_PER_THREAD; d++) {
            qk[j] += __half2float(reg_input[d] * reg_weight[d]);
        }
        #pragma unroll
        for (int mask = (NUM_THREAD_PER_ROW_2 >> 1); mask > 0; mask >>= 1) {
            qk[j] += __shfl_down_sync(0xffffffff, qk[j], mask);
        }
        qk[j] = __expf(qk[j] * __frsqrt_rn(HEAD_DIM));
        local_sum += qk[j];
    }
    bar[3].wait(std::move(token[3]));
    for (int j = 0; j < DEC_TILE; j++) {
        if (cluster_block_id == CLUSTER_SIZE - 1 && warp_id == NUM_WARPS - 1 && lane_id / NUM_THREAD_PER_ROW_2 == 1 && j == DEC_TILE - 1) 
            *(uint4*)(&reg_weight[0]) = *(uint4*)(&local_qkv[2 * HEAD_DIM + input_idx_2]);
        else
            *(uint4*)(&reg_weight[0]) = *(uint4*)(&weight[TMA_LOAD_ONCE_NUM + TMA_LOAD_ONCE_NUM_ATTN + (weight_idx_2 + j) * HEAD_DIM + input_idx_2]);
        #pragma unroll
        for (int d = 0; d < NUM_PER_THREAD; d++) {
            reg_reduce[d] += __float2half(qk[j] * __half2float(reg_weight[d]));
        }
    }
    if (lane_id % NUM_THREAD_PER_ROW_2 == 0) {
        atomicAdd(&reduction[warp_id], local_sum);
    }
    *(uint4*)(&weight[tile_row * HEAD_DIM + tile_col * NUM_PER_THREAD]) = *(uint4*)(&reg_reduce[0]);
    for(int i = 0; i < NUM_PER_THREAD; i++)
        reg_reduce[i] = __float2half(0.0f);
    block.sync();
    if (tid < NUM_WARPS) {
        local_sum = reduction[tid];
    }
    #pragma unroll
    for (int mask = NUM_WARPS / 2; mask > 0; mask >>= 1) {
        local_sum += __shfl_down_sync(0xffffffff, local_sum, mask);
    }
    if(tid == 0) {
        cluster_local_sum = local_sum;
    }
    cluster.sync();
    // DSM Ring-All reduce
    for (int i = 1; i < cluster.num_blocks() - 1; i++) {
        if (tid == 0) {
            local_sum = cluster_local_sum;
            int dst_cta = (cluster_block_id + i) % cluster.num_blocks();
            float* dst_shmem = cluster.map_shared_rank(&cluster_local_sum, dst_cta);
            atomicAdd(dst_shmem, local_sum);
        }
        cluster.sync();
    }
    for (int i = 0; i < NUM_PER_ROW_2; i++) {
        *(uint4*)(&reg_input[0]) = *(uint4*)(&weight[i * HEAD_DIM + tile_col * NUM_PER_THREAD]);
        #pragma unroll
        for (int j = 0; j < NUM_PER_THREAD; j++)
            reg_reduce[j] += reg_input[j];
    }
    if(tid < NUM_THREAD_PER_ROW_2) {
        *(uint4*)(&local_output[tid * NUM_PER_THREAD]) = *(uint4*)(&reg_reduce[0]);
        #pragma unroll
        for (int j = 0; j < NUM_PER_THREAD; j++)
            local_output[tid * NUM_PER_THREAD + j] = __float2half(__half2float(local_output[tid * NUM_PER_THREAD + j]) * __frcp_rn(cluster_local_sum));
    }
    block.sync();

    // DSM Ring-All reduce
    size = HEAD_DIM * sizeof(half);
    for (int i = 1; i < cluster.num_blocks() - 1; i++) {
        if (tid == 0) {
            asm volatile (
                "mbarrier.init.shared::cta.b64 [%0], %1;"
                :
                : "r"(bar_ptr), "r"(1)
            );
            asm volatile (
                "mbarrier.arrive.expect_tx.shared::cta.b64 _, [%0], %1;"
                :
                : "r"(bar_ptr), "r"(size)
            );
        }
        cluster.sync();
        if (tid == 0) {
            uint32_t src_addr = static_cast<uint32_t>(__cvta_generic_to_shared(local_output));
            uint32_t dst_addr = static_cast<uint32_t>(__cvta_generic_to_shared(weight));
            uint32_t dst_cta = (cluster_block_id + i) % cluster.num_blocks();
            uint32_t neighbor_dst_addr;
            asm volatile (
                "mapa.shared::cluster.u32 %0, %1, %2;\n"
                : "=r"(neighbor_dst_addr)
                : "r"(dst_addr), "r"(dst_cta)
            );
            uint32_t neighbor_dst_bar;
            asm volatile (
                "mapa.shared::cluster.u32 %0, %1, %2;\n"
                : "=r"(neighbor_dst_bar)
                : "r"(bar_ptr), "r"(dst_cta)
            );
            asm volatile (
                "cp.async.bulk.shared::cluster.shared::cta.mbarrier::complete_tx::bytes [%0], [%1], %2, [%3];"
                :
                :"r"(neighbor_dst_addr), "r"(src_addr), "r"(size), "r"(neighbor_dst_bar)
                : "memory"
            );
        }
        asm volatile (
            "{\n"
            ".reg .pred                P1;\n"
            "LAB_WAIT:\n"
            "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1;\n"
            "@P1                       bra.uni DONE;\n"
            "bra.uni                   LAB_WAIT;\n"
            "DONE:\n"
            "}\n"
            :: "r"(bar_ptr),
            "r"(0)
        );

        // Add
        if (tid < HEAD_DIM / 2) {
            buffer = *(half2*)(&weight[tid * 2]);
            *(half2*)(&local_output[tid * 2]) = __hadd2(*(half2*)(&local_output[tid * 2]), buffer);
        }
        cluster.sync();
    }
    // if(head_id == 0 && cluster_block_id == 1 && tid == 0)
    //     printf("%f, %f \n", __half2float(local_output[0]), __half2float(local_output[127]));
    // Compute output @ w_o
    // Preload w_o
    if (tid == 0) {
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[0], &tensor_map_weight_o, cluster_block_st_id, head_id * HEAD_DIM, bar[0]);
        token[0] = cuda::device::barrier_arrive_tx(bar[0], 1, TMA_LOAD_ONCE_SIZE);
    } else {
        token[0] = bar[0].arrive();
    }

    for (int id = 1; id < DIM_PER_BLOCK / TMA_LOAD_ONCE; id++) {
        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM], &tensor_map_weight_o, cluster_block_st_id + id * TMA_LOAD_ONCE, head_id * HEAD_DIM, bar[id % 2]);
            token[id % 2] = cuda::device::barrier_arrive_tx(bar[id % 2], 1, TMA_LOAD_ONCE_SIZE);
        } else {
            token[id % 2] = bar[id % 2].arrive();
        }
        bar[(id - 1) % 2].wait(std::move(token[(id - 1) % 2]));
        tmp = 0.0;
        for (int j = 0; j < HEAD_DIM; j+=NUM_PER_ROW_3) {
            *(uint4*)(&reg_input[0]) = *(uint4*)(&local_output[input_idx_3 + j]);
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp += __half2float(reg_input[d] * weight[(id - 1) % 2 * TMA_LOAD_ONCE_NUM + (input_idx_3 + j + d) * TMA_LOAD_ONCE + weight_idx_3]);
            }
        }
        #pragma unroll
        for (int mask = (NUM_THREAD_PER_ROW_3 >> 1); mask > 0; mask >>= 1) {
            tmp += __shfl_down_sync(0xffffffff, tmp, mask);
        }
        if (lane_id % NUM_THREAD_PER_ROW_3 == 0) {
            atomicAdd(&global_reduce[cluster_block_st_id + weight_idx_3 + (id - 1) * TMA_LOAD_ONCE], __float2half(tmp));
        }
    }
    bar[1].wait(std::move(token[1]));
    tmp = 0.0;
    for (int j = 0; j < HEAD_DIM; j+=NUM_PER_ROW_3) {
        *(uint4*)(&reg_input[0]) = *(uint4*)(&local_output[input_idx_3 + j]);
        #pragma unroll
        for (int d = 0; d < NUM_PER_THREAD; d++) {
            tmp += __half2float(reg_input[d] * weight[TMA_LOAD_ONCE_NUM + (input_idx_3 + j + d) * TMA_LOAD_ONCE + weight_idx_3]);
        }
    }
    #pragma unroll
    for (int mask = (NUM_THREAD_PER_ROW_3 >> 1); mask > 0; mask >>= 1) {
        tmp += __shfl_down_sync(0xffffffff, tmp, mask);
    }
    if (lane_id % NUM_THREAD_PER_ROW_3 == 0) {
        atomicAdd(&global_reduce[cluster_block_st_id + weight_idx_3 + ((DIM_PER_BLOCK / TMA_LOAD_ONCE) - 1) * TMA_LOAD_ONCE], __float2half(tmp));
    }
    cluster.sync();

    // Fused residual and RMSNorm
    local_sum = 0.0;
    for (int d = tid * 2; d < DIM_PER_BLOCK; d+=BLOCK_SIZE * 2) { 
        // printf("%f, %f \n", __half2float(input_shmem[d]), __half2float(global_reduce[cluster_block_st_id + d]));
        *(half2*)(&reg_input_norm[0]) = __hadd2(*(half2*)(&input_shmem[d]), *(half2*)(&global_reduce[cluster_block_st_id + d]));
        *(half2*)(&input_shmem[d]) = *(half2*)(&reg_input_norm[0]);
        for (int di = 0; di < 2; di++)
            local_sum += __half2float(reg_input_norm[di] * reg_input_norm[di]);
    }
    #pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        local_sum += __shfl_down_sync(0xffffffff, local_sum, mask);
    }
    if (lane_id == 0){
        reduction[warp_id] = local_sum;
    }
    block.sync(); 
    if (tid < NUM_WARPS){
        local_sum = reduction[tid];
    }
    #pragma unroll
    for (int mask = NUM_WARPS / 2; mask > 0; mask >>= 1) {
        local_sum += __shfl_down_sync(0xffffffff, local_sum, mask);
    } 
    if (tid == 0)
        cluster_local_sum = local_sum;
    cluster.sync();
    // Reduce through DSM
    for (int i = 1; i < cluster.num_blocks() - 1; i++) {
        if (tid == 0) {
            local_sum = cluster_local_sum;
            int dst_cta = (cluster_block_id + i) % cluster.num_blocks();
            float* dst_shmem = cluster.map_shared_rank(&cluster_local_sum, dst_cta);
            atomicAdd(dst_shmem, local_sum);
        }
        cluster.sync();
    }
    rms_rcp = __float2half(1.f / (std::sqrt(cluster_local_sum / float(HIDDEN_DIM)) + eps));
    for (int d = tid * 2; d < DIM_PER_BLOCK; d+=BLOCK_SIZE * 2) { 
        *(half2*)(&reg_input_norm[0]) = *(half2*)(&input_shmem[d]);
        *(half2*)(&reg_input_norm[0]) = __hmul2(*(half2*)(&reg_input_norm[0]), {rms_rcp, rms_rcp});
        *(half2*)(&reg_weight_norm[0]) = *(half2*)(&w_rms_attn[cluster_block_st_id + d]);
        *(half2*)(&input_shmem[d]) = __hmul2(*(half2*)(&reg_input_norm[0]), *(half2*)(&reg_weight_norm[0]));
    }
    block.sync();

    // Compute input @ ffn_gate
    // Preload weight_gate
    if (tid == 0) {
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[0], &tensor_map_weight_gate_up, head_id * FFN_DIM_PER_CLUSTER, cluster_block_st_id, bar[0]);
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[TMA_LOAD_ONCE_NUM_FFN], &tensor_map_weight_gate_up_, head_id * FFN_DIM_PER_CLUSTER + TMA_LOAD_ONCE_MAX, cluster_block_st_id, bar[0]);
        token[0] = cuda::device::barrier_arrive_tx(bar[0], 1, TMA_LOAD_ONCE_SIZE_FFN);
    } else {
        token[0] = bar[0].arrive();
    }

    for (int id = 1; id < DIM_PER_BLOCK / TMA_LOAD_ONCE; id++) {
        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM_FFN_TOTAL], &tensor_map_weight_gate_up, head_id * FFN_DIM_PER_CLUSTER, cluster_block_st_id + id * TMA_LOAD_ONCE, bar[id % 2]);
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN], &tensor_map_weight_gate_up_, head_id * FFN_DIM_PER_CLUSTER + TMA_LOAD_ONCE_MAX, cluster_block_st_id + id * TMA_LOAD_ONCE, bar[id % 2]);
            token[id % 2] = cuda::device::barrier_arrive_tx(bar[id % 2], 1, TMA_LOAD_ONCE_SIZE_FFN);
        } else {
            token[id % 2] = bar[id % 2].arrive();
        }
        bar[(id - 1) % 2].wait(std::move(token[(id - 1) % 2]));
        for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
            *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + (id - 1) * TMA_LOAD_ONCE + i]);
            for (int j = 0; j < TMA_LOAD_ONCE_MAX / HEAD_DIM; j++) {
                #pragma unroll
                for (int d = 0; d < NUM_PER_THREAD; d++) {
                    tmp_ffn[j] += __half2float(reg_input[d] * weight[(id - 1) % 2 * TMA_LOAD_ONCE_NUM_FFN_TOTAL + (input_idx + i + d) * TMA_LOAD_ONCE_MAX + weight_idx + j * HEAD_DIM]);
                }
            }
            for (int j = 0; j < (FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX) / HEAD_DIM; j++) {
                #pragma unroll
                for (int d = 0; d < NUM_PER_THREAD; d++) {
                    tmp_ffn[TMA_LOAD_ONCE_MAX / HEAD_DIM + j] += __half2float(reg_input[d] * weight[(id - 1) % 2 * TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN + (input_idx + i + d) * (FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX) + weight_idx + j * HEAD_DIM]);
                }
            }
        }
    }
    bar[1].wait(std::move(token[1]));
    for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
        *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + ((DIM_PER_BLOCK / TMA_LOAD_ONCE) - 1) * TMA_LOAD_ONCE + i]);
        for (int j = 0; j < TMA_LOAD_ONCE_MAX / HEAD_DIM; j++) {
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp_ffn[j] += __half2float(reg_input[d] * weight[TMA_LOAD_ONCE_NUM_FFN_TOTAL + (input_idx + i + d) * TMA_LOAD_ONCE_MAX + weight_idx + j * HEAD_DIM]);
            }
        }
        for (int j = 0; j < (FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX) / HEAD_DIM; j++) {
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp_ffn[TMA_LOAD_ONCE_MAX / HEAD_DIM + j] += __half2float(reg_input[d] * weight[TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN + (input_idx + i + d) * (FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX) + weight_idx + j * HEAD_DIM]);
            }
        }
    }
    for (int j = 0; j < FFN_DIM_PER_CLUSTER / HEAD_DIM; j++){
        local_qkv[j * HEAD_DIM + warp_id * NUM_ROW_PER_WARP + lane_id / NUM_THREAD_PER_ROW] = __float2half(tmp_ffn[j]);
    }

    // Compute input @ ffn_up
    for (int j = 0; j < FFN_DIM_PER_CLUSTER / HEAD_DIM; j++){
        tmp_ffn[j] = 0.0;
    }
    // Preload weight_up
    if (tid == 0) {
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[0], &tensor_map_weight_gate_up, head_id * FFN_DIM_PER_CLUSTER, HIDDEN_DIM + cluster_block_st_id, bar[0]);
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[TMA_LOAD_ONCE_NUM_FFN], &tensor_map_weight_gate_up_, head_id * FFN_DIM_PER_CLUSTER + TMA_LOAD_ONCE_MAX, HIDDEN_DIM + cluster_block_st_id, bar[0]);
        token[0] = cuda::device::barrier_arrive_tx(bar[0], 1, TMA_LOAD_ONCE_SIZE_FFN);
    } else {
        token[0] = bar[0].arrive();
    }

    for (int id = 1; id < DIM_PER_BLOCK / TMA_LOAD_ONCE; id++) {
        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM_FFN_TOTAL], &tensor_map_weight_gate_up, head_id * FFN_DIM_PER_CLUSTER, HIDDEN_DIM + cluster_block_st_id + id * TMA_LOAD_ONCE, bar[id % 2]);
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN], &tensor_map_weight_gate_up_, head_id * FFN_DIM_PER_CLUSTER + TMA_LOAD_ONCE_MAX, HIDDEN_DIM + cluster_block_st_id + id * TMA_LOAD_ONCE, bar[id % 2]);
            token[id % 2] = cuda::device::barrier_arrive_tx(bar[id % 2], 1, TMA_LOAD_ONCE_SIZE_FFN);
        } else {
            token[id % 2] = bar[id % 2].arrive();
        }
        bar[(id - 1) % 2].wait(std::move(token[(id - 1) % 2]));
        for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
            *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + (id - 1) * TMA_LOAD_ONCE + i]);
            for (int j = 0; j < TMA_LOAD_ONCE_MAX / HEAD_DIM; j++) {
                #pragma unroll
                for (int d = 0; d < NUM_PER_THREAD; d++) {
                    tmp_ffn[j] += __half2float(reg_input[d] * weight[(id - 1) % 2 * TMA_LOAD_ONCE_NUM_FFN_TOTAL + (input_idx + i + d) * TMA_LOAD_ONCE_MAX + weight_idx + j * HEAD_DIM]);
                }
            }
            for (int j = 0; j < (FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX) / HEAD_DIM; j++) {
                #pragma unroll
                for (int d = 0; d < NUM_PER_THREAD; d++) {
                    tmp_ffn[TMA_LOAD_ONCE_MAX / HEAD_DIM + j] += __half2float(reg_input[d] * weight[(id - 1) % 2 * TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN + (input_idx + i + d) * (FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX) + weight_idx + j * HEAD_DIM]);
                }
            }
        }
    }
    bar[1].wait(std::move(token[1]));
    for (int i = 0; i < TMA_LOAD_ONCE; i+=NUM_PER_ROW) { 
        *(uint4*)(&reg_input[0]) = *(uint4*)(&input_shmem[input_idx + ((DIM_PER_BLOCK / TMA_LOAD_ONCE) - 1) * TMA_LOAD_ONCE + i]);
        for (int j = 0; j < TMA_LOAD_ONCE_MAX / HEAD_DIM; j++) {
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp_ffn[j] += __half2float(reg_input[d] * weight[TMA_LOAD_ONCE_NUM_FFN_TOTAL + (input_idx + i + d) * TMA_LOAD_ONCE_MAX + weight_idx + j * HEAD_DIM]);
            }
        }
        for (int j = 0; j < (FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX) / HEAD_DIM; j++) {
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp_ffn[TMA_LOAD_ONCE_MAX / HEAD_DIM + j] += __half2float(reg_input[d] * weight[TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN + (input_idx + i + d) * (FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX) + weight_idx + j * HEAD_DIM]);
            }
        }
    }
    for (int j = 0; j < FFN_DIM_PER_CLUSTER / HEAD_DIM; j++){
        local_qkv[MAX_SMEM_DIM + j * HEAD_DIM + warp_id * NUM_ROW_PER_WARP + lane_id / NUM_THREAD_PER_ROW] = __float2half(tmp_ffn[j]);
    }
    block.sync();

    // DSM All-reduce
    size = FFN_DIM_PER_CLUSTER * 2 * sizeof(half);
    for (int i = 1; i < cluster.num_blocks() - 1; i++) {
        if (tid == 0) {
            asm volatile (
                "mbarrier.init.shared::cta.b64 [%0], %1;"
                :
                : "r"(bar_ptr), "r"(1)
            );
            asm volatile (
                "mbarrier.arrive.expect_tx.shared::cta.b64 _, [%0], %1;"
                :
                : "r"(bar_ptr), "r"(size)
            );
        }
        cluster.sync();
        if (tid == 0) {
            uint32_t src_addr = static_cast<uint32_t>(__cvta_generic_to_shared(local_qkv));
            uint32_t dst_addr = static_cast<uint32_t>(__cvta_generic_to_shared(weight));
            uint32_t dst_cta = (cluster_block_id + i) % cluster.num_blocks();
            uint32_t neighbor_dst_addr;
            asm volatile (
                "mapa.shared::cluster.u32 %0, %1, %2;\n"
                : "=r"(neighbor_dst_addr)
                : "r"(dst_addr), "r"(dst_cta)
            );
            uint32_t neighbor_dst_bar;
            asm volatile (
                "mapa.shared::cluster.u32 %0, %1, %2;\n"
                : "=r"(neighbor_dst_bar)
                : "r"(bar_ptr), "r"(dst_cta)
            );
            asm volatile (
                "cp.async.bulk.shared::cluster.shared::cta.mbarrier::complete_tx::bytes [%0], [%1], %2, [%3];"
                :
                :"r"(neighbor_dst_addr), "r"(src_addr), "r"(size), "r"(neighbor_dst_bar)
                : "memory"
            );
        }
        asm volatile (
            "{\n"
            ".reg .pred                P1;\n"
            "LAB_WAIT:\n"
            "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1;\n"
            "@P1                       bra.uni DONE;\n"
            "bra.uni                   LAB_WAIT;\n"
            "DONE:\n"
            "}\n"
            :: "r"(bar_ptr),
            "r"(0)
        );

        // Add
        if (tid < HEAD_DIM / 2) {
            for (int j = 0; j < FFN_DIM_PER_CLUSTER / HEAD_DIM; j++) {
              buffer = *(half2*)(&weight[j * HEAD_DIM + tid * 2]);
              if (i == cluster.num_blocks() - 2) // ReLU
                  *(half2*)(&local_qkv[j * HEAD_DIM + tid * 2]) = __hmax2(__hadd2(*(half2*)(&local_qkv[j * HEAD_DIM + tid * 2]), buffer), __float22half2_rn({0.0f, 0.0f}));
              else
                  *(half2*)(&local_qkv[j * HEAD_DIM + tid * 2]) = __hadd2(*(half2*)(&local_qkv[j * HEAD_DIM + tid * 2]), buffer);
            }
            for (int j = 0; j < FFN_DIM_PER_CLUSTER / HEAD_DIM; j++) {
                buffer = *(half2*)(&weight[FFN_DIM_PER_CLUSTER + j * HEAD_DIM + tid * 2]);
                *(half2*)(&local_qkv[FFN_DIM_PER_CLUSTER + j * HEAD_DIM + tid * 2]) = __hadd2(*(half2*)(&local_qkv[FFN_DIM_PER_CLUSTER + j * HEAD_DIM + tid * 2]), buffer);
            }
        }
        cluster.sync();
    }
    
    // Compute up_gate mul and down_proj
    if (tid == 0) {
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[0], &tensor_map_weight_down, cluster_block_st_id, head_id * FFN_DIM_PER_CLUSTER, bar[0]);
        cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[TMA_LOAD_ONCE_NUM_FFN], &tensor_map_weight_down_, cluster_block_st_id, head_id * FFN_DIM_PER_CLUSTER + TMA_LOAD_ONCE_MAX, bar[0]);
        token[0] = cuda::device::barrier_arrive_tx(bar[0], 1, TMA_LOAD_ONCE_SIZE_FFN);
    } else {
        token[0] = bar[0].arrive();
    }

    for (int id = 1; id < DIM_PER_BLOCK / TMA_LOAD_ONCE; id++) {
        if (tid == 0) {
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM_FFN_TOTAL], &tensor_map_weight_down, cluster_block_st_id + id * TMA_LOAD_ONCE, head_id * FFN_DIM_PER_CLUSTER, bar[id % 2]);
            cde::cp_async_bulk_tensor_2d_global_to_shared(&weight[(id % 2) * TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN], &tensor_map_weight_down_, cluster_block_st_id + id * TMA_LOAD_ONCE, head_id * FFN_DIM_PER_CLUSTER + TMA_LOAD_ONCE_MAX, bar[id % 2]);
            token[id % 2] = cuda::device::barrier_arrive_tx(bar[id % 2], 1, TMA_LOAD_ONCE_SIZE_FFN);
        } else {
            token[id % 2] = bar[id % 2].arrive();
        }
        bar[(id - 1) % 2].wait(std::move(token[(id - 1) % 2]));
        tmp = 0.0;
        for (int j = 0; j < TMA_LOAD_ONCE_MAX; j+=NUM_PER_ROW_3) {
            *(uint4*)(&reg_input[0]) = *(uint4*)(&local_qkv[input_idx_3 + j]);
            *(uint4*)(&reg_reduce[0]) = *(uint4*)(&local_qkv[MAX_SMEM_DIM + input_idx_3 + j]);
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp += __half2float(reg_input[d] * reg_reduce[d] * weight[(id - 1) % 2 * TMA_LOAD_ONCE_NUM_FFN_TOTAL + (input_idx_3 + j + d) * TMA_LOAD_ONCE + weight_idx_3]);
            }
        }
        for (int j = 0; j < FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX; j+=NUM_PER_ROW_3) {
            *(uint4*)(&reg_input[0]) = *(uint4*)(&local_qkv[input_idx_3 + TMA_LOAD_ONCE_MAX + j]);
            *(uint4*)(&reg_reduce[0]) = *(uint4*)(&local_qkv[MAX_SMEM_DIM + input_idx_3 + TMA_LOAD_ONCE_MAX + j]);
            #pragma unroll
            for (int d = 0; d < NUM_PER_THREAD; d++) {
                tmp += __half2float(reg_input[d] * reg_reduce[d] * weight[(id - 1) % 2 * TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN + (input_idx_3 + j + d) * TMA_LOAD_ONCE + weight_idx_3]);
            }
        }
        #pragma unroll
        for (int mask = (NUM_THREAD_PER_ROW_3 >> 1); mask > 0; mask >>= 1) {
            tmp += __shfl_down_sync(0xffffffff, tmp, mask);
        }
        if (lane_id % NUM_THREAD_PER_ROW_3 == 0) {
            atomicAdd(&output[cluster_block_st_id + weight_idx_3 + (id - 1) * TMA_LOAD_ONCE], __float2half(tmp));
        }
    }
    bar[1].wait(std::move(token[1]));
    tmp = 0.0;
    for (int j = 0; j < TMA_LOAD_ONCE_MAX; j+=NUM_PER_ROW_3) {
        *(uint4*)(&reg_input[0]) = *(uint4*)(&local_qkv[input_idx_3 + j]);
        *(uint4*)(&reg_reduce[0]) = *(uint4*)(&local_qkv[MAX_SMEM_DIM + input_idx_3 + j]);
        #pragma unroll
        for (int d = 0; d < NUM_PER_THREAD; d++) {
            tmp += __half2float(reg_input[d] * reg_reduce[d] * weight[TMA_LOAD_ONCE_NUM_FFN_TOTAL + (input_idx_3 + j + d) * TMA_LOAD_ONCE + weight_idx_3]);
        }
    }
    for (int j = 0; j < FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX; j+=NUM_PER_ROW_3) {
        *(uint4*)(&reg_input[0]) = *(uint4*)(&local_qkv[input_idx_3 + TMA_LOAD_ONCE_MAX + j]);
        *(uint4*)(&reg_reduce[0]) = *(uint4*)(&local_qkv[MAX_SMEM_DIM + input_idx_3 + TMA_LOAD_ONCE_MAX + j]);
        #pragma unroll
        for (int d = 0; d < NUM_PER_THREAD; d++) {
            tmp += __half2float(reg_input[d] * reg_reduce[d] * weight[TMA_LOAD_ONCE_NUM_FFN_TOTAL + TMA_LOAD_ONCE_NUM_FFN + (input_idx_3 + j + d) * TMA_LOAD_ONCE + weight_idx_3]);
        }
    }
    #pragma unroll
    for (int mask = (NUM_THREAD_PER_ROW_3 >> 1); mask > 0; mask >>= 1) {
        tmp += __shfl_down_sync(0xffffffff, tmp, mask);
    }
    if (lane_id % NUM_THREAD_PER_ROW_3 == 0) {
        atomicAdd(&output[cluster_block_st_id + weight_idx_3 + ((DIM_PER_BLOCK / TMA_LOAD_ONCE) - 1) * TMA_LOAD_ONCE], __float2half(tmp));
    }
}

int main(int argc, char** argv) {
    hipFuncSetAttribute(reinterpret_cast<const void*>(single_decode), cudaFuncAttributeNonPortableClusterSizeAllowed, 16);
    uint32_t max_shmem_size = 0;
    hipFuncSetAttribute(reinterpret_cast<const void*>(single_decode), hipFuncAttributeMaxDynamicSharedMemorySize, max_shmem_size);
    half *h_input, *d_input;
    half *h_k_cache, *d_k_cache;
    half *h_v_cache, *d_v_cache;
    half *h_w_qkv, *d_w_qkv;
    half *h_w_o, *d_w_o;
    half *h_ffn_gate_up, *d_ffn_gate_up;
    half *h_ffn_down, *d_ffn_down;
    half *h_rms_input, *d_rms_input;
    half *h_rms_attn, *d_rms_attn;
    float *h_cos, *d_cos;
    float *h_sin, *d_sin;
    h_input = new half[1 * HIDDEN_DIM];
    h_w_qkv = new half[3 * HIDDEN_DIM * HIDDEN_DIM];
    h_w_o = new half[HIDDEN_DIM * HIDDEN_DIM];
    h_k_cache = new half[SEQ_LEN * HEAD_NUM * HEAD_DIM];
    h_v_cache = new half[SEQ_LEN * HEAD_NUM * HEAD_DIM];
    h_ffn_gate_up = new half[2 * HIDDEN_DIM * FFN_DIM];
    h_ffn_down = new half[FFN_DIM * HIDDEN_DIM];
    h_rms_input = new half[HIDDEN_DIM];
    h_rms_attn = new half[HIDDEN_DIM];
    h_cos = new float[HEAD_DIM];
    h_sin = new float[HEAD_DIM];

    fill_matrix(h_input, 1 * HIDDEN_DIM);
    fill_matrix(h_w_qkv, 3 * HIDDEN_DIM * HIDDEN_DIM);
    fill_matrix(h_w_o, HIDDEN_DIM * HIDDEN_DIM);
    fill_matrix(h_k_cache, SEQ_LEN * HEAD_NUM * HEAD_DIM);
    fill_matrix(h_v_cache, SEQ_LEN * HEAD_NUM * HEAD_DIM);
    fill_matrix(h_ffn_gate_up, 2 * HIDDEN_DIM * FFN_DIM);
    fill_matrix(h_ffn_down, FFN_DIM * HIDDEN_DIM);
    fill_matrix(h_rms_input, HIDDEN_DIM);
    fill_matrix(h_rms_attn, HIDDEN_DIM);

    // Init cos, sin used in RoPE
    int encode_point_offset = 0;
    float rope_scale = 1;
    float rope_theta = 500000;
    for (int j = 0; j < HEAD_DIM; j++) {
        float inv_freq =(encode_point_offset / rope_scale) / (std::pow(rope_theta, float(2 * (j % (HEAD_DIM / 2))) / float(HEAD_DIM)));
        h_cos[j] = std::cos(inv_freq);
        h_sin[j] = std::sin(inv_freq);
    }

    hipMalloc(reinterpret_cast<void**>(&d_input), sizeof(half) * 1 * HIDDEN_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_w_qkv), sizeof(half) * 3 * HIDDEN_DIM * HIDDEN_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_w_o), sizeof(half) * HIDDEN_DIM * HIDDEN_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_k_cache), sizeof(half) * SEQ_LEN * HEAD_NUM * HEAD_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_v_cache), sizeof(half) * SEQ_LEN * HEAD_NUM * HEAD_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_ffn_gate_up), sizeof(half) * 2 * HIDDEN_DIM * FFN_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_ffn_down), sizeof(half) * FFN_DIM * HIDDEN_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_rms_input), sizeof(half) * HIDDEN_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_rms_attn), sizeof(half) * HIDDEN_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_cos), sizeof(float) * HEAD_DIM);
    hipMalloc(reinterpret_cast<void**>(&d_sin), sizeof(float) * HEAD_DIM);

    hipMemcpy(reinterpret_cast<void*>(d_input), h_input, sizeof(half) * 1 * HIDDEN_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_w_qkv), h_w_qkv, sizeof(half) * 3 * HIDDEN_DIM * HIDDEN_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_w_o), h_w_o, sizeof(half) * HIDDEN_DIM * HIDDEN_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_k_cache), h_k_cache, sizeof(half) * SEQ_LEN * HEAD_NUM * HEAD_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_v_cache), h_v_cache, sizeof(half) * SEQ_LEN * HEAD_NUM * HEAD_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_ffn_gate_up), h_ffn_gate_up, sizeof(half) * 2 * HIDDEN_DIM * FFN_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_ffn_down), h_ffn_down, sizeof(half) * FFN_DIM * HIDDEN_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_rms_input), h_rms_input, sizeof(half) * HIDDEN_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_rms_attn), h_rms_attn, sizeof(half) * HIDDEN_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_cos), h_cos, sizeof(float) * HEAD_DIM, hipMemcpyHostToDevice);
    hipMemcpy(reinterpret_cast<void*>(d_sin), h_sin, sizeof(float) * HEAD_DIM, hipMemcpyHostToDevice);

    half* h_output, *d_output;
    h_output = new half[1 * HIDDEN_DIM];
    hipMalloc(reinterpret_cast<void**>(&d_output), sizeof(half) * 1 * HIDDEN_DIM);
    
    half *global_reduce;
    hipMalloc(reinterpret_cast<void**>(&global_reduce), sizeof(half) * HIDDEN_DIM);
    
    CUtensorMap tensor_map_weight{};
    CUtensorMap tensor_map_k_cache{};
    CUtensorMap tensor_map_v_cache{};
    CUtensorMap tensor_map_weight_o{};
    CUtensorMap tensor_map_weight_gate_up{};
    CUtensorMap tensor_map_weight_gate_up_{};
    CUtensorMap tensor_map_weight_down{};
    CUtensorMap tensor_map_weight_down_{};
    // rank is the number of dimensions of the array.
    constexpr uint32_t rank = 2;
    uint64_t size[rank] = {HIDDEN_DIM, 3 * HIDDEN_DIM};
    // The stride is the number of bytes to traverse from the first element of one row to the next.
    // It must be a multiple of 16.
    uint64_t stride[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    // The box_size is the size of the shared memory buffer that is used as the
    // destination of a TMA transfer.
    uint32_t box_size[rank] = {HEAD_DIM, TMA_LOAD_ONCE};
    // The distance between elements in units of sizeof(element). A stride of 2
    // can be used to load only the real component of a complex-valued tensor, for instance.
    uint32_t elem_stride[rank] = {1, 1};

    // Create the tensor descriptor.
    hipError_t res = cuTensorMapEncodeTiled(
        &tensor_map_weight,                // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       // cuuint32_t tensorRank,
        d_w_qkv,                 // void *globalAddress,
        size,                       // const cuuint64_t *globalDim,
        stride,                     // const cuuint64_t *globalStrides,
        box_size,                   // const cuuint32_t *boxDim,
        elem_stride,                // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_k_cache[rank] = {HIDDEN_DIM, SEQ_LEN};
    uint64_t stride_k_cache[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_k_cache[rank] = {HEAD_DIM, TMA_LOAD_ONCE / 2};
    uint32_t elem_stride_k_cache[rank] = {1, 1};

    // Create the tensor descriptor.
    hipError_t res_k_cache = cuTensorMapEncodeTiled(
        &tensor_map_k_cache,                // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       // cuuint32_t tensorRank,
        d_k_cache,                 // void *globalAddress,
        size_k_cache,                       // const cuuint64_t *globalDim,
        stride_k_cache,                     // const cuuint64_t *globalStrides,
        box_size_k_cache,                   // const cuuint32_t *boxDim,
        elem_stride_k_cache,                // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_v_cache[rank] = {HIDDEN_DIM, SEQ_LEN};
    uint64_t stride_v_cache[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_v_cache[rank] = {HEAD_DIM, TMA_LOAD_ONCE / 2};
    uint32_t elem_stride_v_cache[rank] = {1, 1};

    // Create the tensor descriptor.
    hipError_t res_v_cache = cuTensorMapEncodeTiled(
        &tensor_map_v_cache,                // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       // cuuint32_t tensorRank,
        d_v_cache,                 // void *globalAddress,
        size_v_cache,                       // const cuuint64_t *globalDim,
        stride_v_cache,                     // const cuuint64_t *globalStrides,
        box_size_v_cache,                   // const cuuint32_t *boxDim,
        elem_stride_v_cache,                // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_o[rank] = {HIDDEN_DIM, HIDDEN_DIM};
    uint64_t stride_weight_o[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_weight_o[rank] = {TMA_LOAD_ONCE, HEAD_DIM};
    uint32_t elem_stride_weight_o[rank] = {1, 1};

    // Create the tensor descriptor.
    hipError_t res_weight_o = cuTensorMapEncodeTiled(
        &tensor_map_weight_o,                // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       // cuuint32_t tensorRank,
        d_w_o,                 // void *globalAddress,
        size_weight_o,                       // const cuuint64_t *globalDim,
        stride_weight_o,                     // const cuuint64_t *globalStrides,
        box_size_weight_o,                   // const cuuint32_t *boxDim,
        elem_stride_weight_o,                // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_gate_up[rank] = {FFN_DIM, 2 * HIDDEN_DIM};
    uint64_t stride_weight_gate_up[rank - 1] = {FFN_DIM * sizeof(half)};
    uint32_t box_size_weight_gate_up[rank] = {TMA_LOAD_ONCE_MAX, TMA_LOAD_ONCE};
    uint32_t elem_stride_weight_gate_up[rank] = {1, 1};

    // Create the tensor descriptor.
    hipError_t res_weight_gate_up = cuTensorMapEncodeTiled(
        &tensor_map_weight_gate_up,                // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       // cuuint32_t tensorRank,
        d_ffn_gate_up,                 // void *globalAddress,
        size_weight_gate_up,                       // const cuuint64_t *globalDim,
        stride_weight_gate_up,                     // const cuuint64_t *globalStrides,
        box_size_weight_gate_up,                   // const cuuint32_t *boxDim,
        elem_stride_weight_gate_up,                // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_gate_up_[rank] = {FFN_DIM, 2 * HIDDEN_DIM};
    uint64_t stride_weight_gate_up_[rank - 1] = {FFN_DIM * sizeof(half)};
    uint32_t box_size_weight_gate_up_[rank] = {FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX, TMA_LOAD_ONCE};
    uint32_t elem_stride_weight_gate_up_[rank] = {1, 1};

    // Create the tensor descriptor.
    hipError_t res_weight_gate_up_ = cuTensorMapEncodeTiled(
        &tensor_map_weight_gate_up_,                // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       // cuuint32_t tensorRank,
        d_ffn_gate_up,                 // void *globalAddress,
        size_weight_gate_up_,                       // const cuuint64_t *globalDim,
        stride_weight_gate_up_,                     // const cuuint64_t *globalStrides,
        box_size_weight_gate_up_,                   // const cuuint32_t *boxDim,
        elem_stride_weight_gate_up_,                // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_down[rank] = {HIDDEN_DIM, FFN_DIM};
    uint64_t stride_weight_down[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_weight_down[rank] = {TMA_LOAD_ONCE, TMA_LOAD_ONCE_MAX};
    uint32_t elem_stride_weight_down[rank] = {1, 1};

    // Create the tensor descriptor.
    hipError_t res_weight_down = cuTensorMapEncodeTiled(
        &tensor_map_weight_down,                // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       // cuuint32_t tensorRank,
        d_ffn_down,                 // void *globalAddress,
        size_weight_down,                       // const cuuint64_t *globalDim,
        stride_weight_down,                     // const cuuint64_t *globalStrides,
        box_size_weight_down,                   // const cuuint32_t *boxDim,
        elem_stride_weight_down,                // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_down_[rank] = {HIDDEN_DIM, FFN_DIM};
    uint64_t stride_weight_down_[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_weight_down_[rank] = {TMA_LOAD_ONCE, FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX};
    uint32_t elem_stride_weight_down_[rank] = {1, 1};

    // Create the tensor descriptor.
    hipError_t res_weight_down_ = cuTensorMapEncodeTiled(
        &tensor_map_weight_down_,                // CUtensorMap *tensorMap,
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       // cuuint32_t tensorRank,
        d_ffn_down,                 // void *globalAddress,
        size_weight_down_,                       // const cuuint64_t *globalDim,
        stride_weight_down_,                     // const cuuint64_t *globalStrides,
        box_size_weight_down_,                   // const cuuint32_t *boxDim,
        elem_stride_weight_down_,                // const cuuint32_t *elementStrides,
        // Interleave patterns can be used to accelerate loading of values that
        // are less than 4 bytes long.
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        // Swizzling can be used to avoid shared memory bank conflicts.
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        // L2 Promotion can be used to widen the effect of a cache-policy to a wider
        // set of L2 cache lines.
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        // Any element that is outside of bounds will be set to zero by the TMA transfer.
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    dim3 grid(HEAD_NUM * CLUSTER_SIZE); 
    dim3 block(BLOCK_SIZE);

    int wmup = 1;
    int test = 0;
    for (int i = 0; i < wmup; i++) {
        single_decode<<<grid, block, max_shmem_size>>>(
            d_output,
            d_input,
            global_reduce,
            d_rms_input,
            d_rms_attn,
            d_cos,
            d_sin,
            tensor_map_weight,
            tensor_map_k_cache,
            tensor_map_v_cache,
            tensor_map_weight_o,
            tensor_map_weight_gate_up,
            tensor_map_weight_gate_up_,
            tensor_map_weight_down,
            tensor_map_weight_down_
        );
    }
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    hipEvent_t st, ed;
    hipEventCreate(&st);
    hipEventCreate(&ed);
    hipEventRecord(st);
    for (int i = 0; i < test; i++) {
        single_decode<<<grid, block, max_shmem_size>>>(
            d_output,
            d_input,
            global_reduce,
            d_rms_input,
            d_rms_attn,
            d_cos,
            d_sin,
            tensor_map_weight,
            tensor_map_k_cache,
            tensor_map_v_cache,
            tensor_map_weight_o,
            tensor_map_weight_gate_up,
            tensor_map_weight_gate_up_,
            tensor_map_weight_down,
            tensor_map_weight_down_
        );
    }
    hipEventRecord(ed);
    hipEventSynchronize(ed);
    float ms;
    hipEventElapsedTime(&ms, st, ed);
    std::cout << "Latency: " << ms / test * 1e3 << " us" << std::endl;
    hipMemcpy(h_output, reinterpret_cast<void*>(d_output), sizeof(half) * 1 * HIDDEN_DIM, hipMemcpyDeviceToHost);
    // for (int i = 0; i < HIDDEN_DIM; i++)
    //     printf("%f, ", __half2float(h_output[i]));
    return 0;
}