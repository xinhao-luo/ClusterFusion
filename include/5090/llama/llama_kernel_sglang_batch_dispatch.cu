#include "hip/hip_runtime.h"
#include "kernel_sglang.cuh"
#include <torch/extension.h>

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor> llama_decoder_layer_sglang_sm120(
    torch::Tensor input,
    torch::Tensor residual,
    torch::Tensor weight_qkv,
    torch::Tensor weight_o,
    torch::Tensor k_cache,
    torch::Tensor v_cache,
    torch::Tensor rms_input_weight,
    float eps,
    torch::Tensor cos,
    torch::Tensor sin
) 