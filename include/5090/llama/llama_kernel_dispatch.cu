#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <torch/extension.h>

torch::Tensor llama_decoder_layer_sm120(
    torch::Tensor input,
    torch::Tensor weight_qkv,
    torch::Tensor weight_o,
    torch::Tensor k_cache,
    torch::Tensor v_cache,
    torch::Tensor gate_up_proj_weight,
    torch::Tensor down_proj_weight,
    torch::Tensor rms_input_weight,
    torch::Tensor rms_attn_weight,
    torch::Tensor cos,
    torch::Tensor sin
) 
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(LlamaDecoderLayerKernel), cudaFuncAttributeNonPortableClusterSizeAllowed, 1);
    uint32_t max_shmem_size = ((((DIM_PER_BLOCK * sizeof(half) + 2 * DIM_BLOCK_REDUCE * sizeof(float) + 127) & ~127) +  2 * TMA_LOAD_ONCE * MAX_SMEM_DIM * sizeof(half) + 127) & ~127) + 3 * HEAD_DIM * sizeof(half);
    hipFuncSetAttribute(reinterpret_cast<const void*>(LlamaDecoderLayerKernel), hipFuncAttributeMaxDynamicSharedMemorySize, max_shmem_size);
    auto options = torch::TensorOptions().dtype(torch::kFloat16).device(torch::kCUDA, 0);
    torch::Tensor o = torch::full({1, HIDDEN_DIM}, 0, options);
    half* o_ptr = reinterpret_cast<half*>(o.data_ptr<at::Half>());
    half *reduce_workspace;
    hipMalloc(reinterpret_cast<void**>(&reduce_workspace), sizeof(half) * 1 * HIDDEN_DIM);

    half* input_ptr = reinterpret_cast<half*>(input.data_ptr<at::Half>());
    half* weight_qkv_ptr = reinterpret_cast<half*>(weight_qkv.data_ptr<at::Half>());
    half* weight_o_ptr = reinterpret_cast<half*>(weight_o.data_ptr<at::Half>());
    half* k_cache_ptr = reinterpret_cast<half*>(k_cache.data_ptr<at::Half>());
    half* v_cache_ptr = reinterpret_cast<half*>(v_cache.data_ptr<at::Half>());
    half* gate_up_proj_weight_ptr = reinterpret_cast<half*>(gate_up_proj_weight.data_ptr<at::Half>());
    half* down_proj_weight_ptr = reinterpret_cast<half*>(down_proj_weight.data_ptr<at::Half>());
    half* rms_input_weight_ptr = reinterpret_cast<half*>(rms_input_weight.data_ptr<at::Half>());
    half* rms_attn_weight_ptr = reinterpret_cast<half*>(rms_attn_weight.data_ptr<at::Half>());
    float* cos_ptr = reinterpret_cast<float*>(cos.data_ptr<float>());
    float* sin_ptr = reinterpret_cast<float*>(sin.data_ptr<float>());
    
    CUtensorMap tensor_map_weight{};
    CUtensorMap tensor_map_k_cache{};
    CUtensorMap tensor_map_v_cache{};
    CUtensorMap tensor_map_weight_o{};
    CUtensorMap tensor_map_weight_gate_up{};
    CUtensorMap tensor_map_weight_gate_up_{};
    CUtensorMap tensor_map_weight_down{};
    CUtensorMap tensor_map_weight_down_{};
    
    constexpr uint32_t rank = 2;
    uint64_t size[rank] = {HIDDEN_DIM, 3 * HIDDEN_DIM};
    uint64_t stride[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size[rank] = {HEAD_DIM, TMA_LOAD_ONCE};
    uint32_t elem_stride[rank] = {1, 1};
    hipError_t res = cuTensorMapEncodeTiled(
        &tensor_map_weight,                
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       
        weight_qkv_ptr,                 
        size,                       
        stride,                     
        box_size,                   
        elem_stride,                
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_k_cache[rank] = {HIDDEN_DIM, SEQ_LEN};
    uint64_t stride_k_cache[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_k_cache[rank] = {HEAD_DIM, TMA_LOAD_ONCE / 2};
    uint32_t elem_stride_k_cache[rank] = {1, 1};

    hipError_t res_k_cache = cuTensorMapEncodeTiled(
        &tensor_map_k_cache,               
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                      
        k_cache_ptr,                
        size_k_cache,                      
        stride_k_cache,                     
        box_size_k_cache,                   
        elem_stride_k_cache,                
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_v_cache[rank] = {HIDDEN_DIM, SEQ_LEN};
    uint64_t stride_v_cache[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_v_cache[rank] = {HEAD_DIM, TMA_LOAD_ONCE / 2};
    uint32_t elem_stride_v_cache[rank] = {1, 1};

    hipError_t res_v_cache = cuTensorMapEncodeTiled(
        &tensor_map_v_cache,               
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                      
        v_cache_ptr,                
        size_v_cache,                      
        stride_v_cache,                     
        box_size_v_cache,                   
        elem_stride_v_cache,                
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );


    uint64_t size_weight_o[rank] = {HIDDEN_DIM, HIDDEN_DIM};
    uint64_t stride_weight_o[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_weight_o[rank] = {TMA_LOAD_ONCE, HEAD_DIM};
    uint32_t elem_stride_weight_o[rank] = {1, 1};
    hipError_t res_weight_o = cuTensorMapEncodeTiled(
        &tensor_map_weight_o,                
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       
        weight_o_ptr,                 
        size_weight_o,                       
        stride_weight_o,                     
        box_size_weight_o,                   
        elem_stride_weight_o,                
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_gate_up[rank] = {FFN_DIM, 2 * HIDDEN_DIM};
    uint64_t stride_weight_gate_up[rank - 1] = {FFN_DIM * sizeof(half)};
    uint32_t box_size_weight_gate_up[rank] = {TMA_LOAD_ONCE_MAX, TMA_LOAD_ONCE};
    uint32_t elem_stride_weight_gate_up[rank] = {1, 1};
    hipError_t res_weight_gate_up = cuTensorMapEncodeTiled(
        &tensor_map_weight_gate_up,                
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       
        gate_up_proj_weight_ptr,                 
        size_weight_gate_up,                       
        stride_weight_gate_up,                     
        box_size_weight_gate_up,                   
        elem_stride_weight_gate_up,                
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_gate_up_[rank] = {FFN_DIM, 2 * HIDDEN_DIM};
    uint64_t stride_weight_gate_up_[rank - 1] = {FFN_DIM * sizeof(half)};
    uint32_t box_size_weight_gate_up_[rank] = {FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX, TMA_LOAD_ONCE};
    uint32_t elem_stride_weight_gate_up_[rank] = {1, 1};
    hipError_t res_weight_gate_up_ = cuTensorMapEncodeTiled(
        &tensor_map_weight_gate_up_,                
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       
        gate_up_proj_weight_ptr,                 
        size_weight_gate_up_,                       
        stride_weight_gate_up_,                     
        box_size_weight_gate_up_,                 
        elem_stride_weight_gate_up_,               
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_down[rank] = {HIDDEN_DIM, FFN_DIM};
    uint64_t stride_weight_down[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_weight_down[rank] = {TMA_LOAD_ONCE, TMA_LOAD_ONCE_MAX};
    uint32_t elem_stride_weight_down[rank] = {1, 1};
    hipError_t res_weight_down = cuTensorMapEncodeTiled(
        &tensor_map_weight_down,                
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                       
        down_proj_weight_ptr,                
        size_weight_down,                      
        stride_weight_down,                   
        box_size_weight_down,                 
        elem_stride_weight_down,                
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    uint64_t size_weight_down_[rank] = {HIDDEN_DIM, FFN_DIM};
    uint64_t stride_weight_down_[rank - 1] = {HIDDEN_DIM * sizeof(half)};
    uint32_t box_size_weight_down_[rank] = {TMA_LOAD_ONCE, FFN_DIM_PER_CLUSTER - TMA_LOAD_ONCE_MAX};
    uint32_t elem_stride_weight_down_[rank] = {1, 1};
    hipError_t res_weight_down_ = cuTensorMapEncodeTiled(
        &tensor_map_weight_down_,             
        CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_FLOAT16,
        rank,                     
        down_proj_weight_ptr,               
        size_weight_down_,                      
        stride_weight_down_,                    
        box_size_weight_down_,                   
        elem_stride_weight_down_,               
        CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
        CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE,
        CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
    );

    dim3 grid(HEAD_NUM * CLUSTER_SIZE); 
    dim3 block(BLOCK_SIZE);

    hipDeviceSynchronize();
    LlamaDecoderLayerKernel<<<grid, block, max_shmem_size>>>(
        o_ptr,
        input_ptr,
        reduce_workspace,
        rms_input_weight_ptr,
        rms_attn_weight_ptr,
        cos_ptr,
        sin_ptr,
        tensor_map_weight,
        tensor_map_k_cache,
        tensor_map_v_cache,
        tensor_map_weight_o,
        tensor_map_weight_gate_up,
        tensor_map_weight_gate_up_,
        tensor_map_weight_down,
        tensor_map_weight_down_
    );
    hipDeviceSynchronize();
    hipFree(reduce_workspace);
    return o;
}